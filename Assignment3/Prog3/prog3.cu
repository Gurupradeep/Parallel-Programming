#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 100

__global__ void sum(int a[][N], int b[][N], int c[][N]) {
	int row_index = blockDim.y * blockIdx.y + threadIdx.y;
	int col_index = blockDim.x * blockIdx.x + threadIdx.x;
	if (row_index < N && col_index < N) {
		c[row_index][col_index] = a[row_index][col_index] + b[row_index][col_index];
	}
}

void handle_error(hipError_t error) {
	if (error != hipSuccess) {
		std::cout << "Cuda Error. Exiting...";	
		exit(0);
	}
}

int main() {
	int a[N][N], b[N][N], c[N][N];
	int (*device_a)[N], (*device_b)[N], (*device_c)[N];
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a[i][j] = i + j;
			b[i][j] = 2*i + j;
		}
	}
	handle_error(hipMalloc((void **)&device_a, N * N * sizeof(int)));
	handle_error(hipMalloc((void **)&device_b, N * N * sizeof(int)));
	handle_error(hipMalloc((void **)&device_c, N * N * sizeof(int)));
	
	hipMemcpy(device_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
	
	dim3 thread_size(8, 8);
	dim3 block_grid_size(ceil(N/8.0), ceil(N/8.0));
	
	sum<<<block_grid_size, thread_size>>>(device_a, device_b, device_c);
	
	hipMemcpy(c, device_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			assert(c[i][j] == a[i][j] + b[i][j]);
		}
	}
	std::cout << "Successful..";
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	return 0;
}

