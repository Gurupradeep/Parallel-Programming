#include <iostream>
#include <math.h> 
#include <hip/hip_runtime.h>
#include <assert.h>

#define N 65564

__global__ void sum(float *a, float *b, float *c) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N) {
		c[index] = a[index] + b[index];
	}
}

void handle_error(hipError_t error) {
	if (error != hipSuccess) {
		std::cout << "Cuda Error. Exiting..";
		exit (0);
	}
}

int main() {
	float a[N], b[N], c[N];
	float *device_a, *device_b, *device_c;
	for (int i = 0; i < N; i++) {
		a[i] = (i+1) * 1.0 / 2;
		b[i] = (i+3) * 1.0 / 3;
	}
	handle_error(hipMalloc((void **) &device_a, N * sizeof(float)));
	handle_error(hipMalloc((void **) &device_b, N * sizeof(float)));
	handle_error(hipMalloc((void **) &device_c, N * sizeof(float)));

	hipMemcpy(device_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_b, b, N * sizeof(float), hipMemcpyHostToDevice);
	sum<<<ceil(N/1024.0), 1024>>>(device_a, device_b, device_c);
	hipMemcpy(c, device_c, N * sizeof(N), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		assert(c[i] == a[i] + b[i]);
	}
	std::cout << "Successful.\n";
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	return 0;
}

